/*
Copyright (c) 2014, Romain Dolbeau, unless otherwise noted.
No claims is made upon work by others.

For the work by Romain Dolbeau:
All rights reserved

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of the <organization> nor the
      names of its contributors may be used to endorse or promote products
      derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

For the work by others:
See the indicated reference for the relevant license.
*/

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>


extern "C" {
unsigned int sleep(unsigned int seconds);
}
#include <cryptopp/aes.h>
#include <cryptopp/modes.h>
#include <cryptopp/gcm.h>
#include <cryptopp/filters.h>
#include <cryptopp/channels.h>

#include <openssl/evp.h>

#define SIZE ((1024*1024*128))

#include "aes_common.h"
static inline void print16c(const uint8_t* buf) {
  uint64_t i;
  for(i = 0 ; i < 16 ; i++) {
    printf("%02x ", buf[i]);
    if (i%4==3)
      printf(" ");
  }
  printf("\n");
}
#define CHECK(K)                                                        \
  do {                                                                  \
    err = K;                                                            \
    if (err) {                                                          \
      fprintf(stderr, "Oups, "#K" failed with %d (%s)\n", err, hipGetErrorString(err)); \
      fflush(stderr);                                                   \
      exit(-2);                                                         \
    } } while (0)



texture<unsigned short, 1, hipReadModeElementType> tFSbSq;

#include "aes_scalar.h"
#include "aes_gcm.h"
#define GPU_CREATE_ALL
#include "aes_gpu.h"


static inline void test_cuda_gcm(const uint8_t *in, uint8_t *out3, const uint8_t* out2, const uint32_t *aes_edrk,
                   const uint32_t *FT0, const uint32_t *FT1, const uint32_t *FT2, const uint32_t *FT3,
                   const uint32_t* IV,
                   aes_encrypt_cuda_proto fct, const int tbp) {
  uint64_t i, j;
  double t_[12], t0, t1;
  int count;
  int tc = 0;
  dim3 g, b;
  uint32_t *gin, *gout;
  uint32_t *gFT0 = NULL, *gFT1 = NULL, *gFT2 = NULL, *gFT3 = NULL;
  uint32_t *gFSb;
  uint32_t *gaes_edrk;
  uint32_t *gIV = NULL;
  uint32_t n_aes_block = (SIZE+15)/16;
  hipError_t err;
  uint8_t accum[16];
  uint32_t temp[4];
  uint8_t H[16];
  uint8_t T[16];
  
  t_[tc++] = wallclock();
  CHECK(hipMalloc((void**)&gin,(size_t)SIZE));
  CHECK(hipMalloc((void**)&gout,(size_t)SIZE));
  if (FT0 != NULL)
    CHECK(hipMalloc((void**)&gFT0,(size_t)1024));
  if (FT1 != NULL)
    CHECK(hipMalloc((void**)&gFT1,(size_t)1024));
  if (FT2 != NULL)
    CHECK(hipMalloc((void**)&gFT2,(size_t)1024));
  if (FT3 != NULL)
    CHECK(hipMalloc((void**)&gFT3,(size_t)1024));
  CHECK(hipMalloc((void**)&gFSb,(size_t)1024));
  CHECK(hipMalloc((void**)&gaes_edrk,(size_t)256));
  if (IV != NULL)
    CHECK(hipMalloc((void**)&gIV,(size_t)16));
  t_[tc++] = wallclock();
  CHECK(hipMemcpy(gin, in, SIZE, hipMemcpyHostToDevice));
  if (FT0 != NULL)
    CHECK(hipMemcpy(gFT0, FT0, 1024, hipMemcpyHostToDevice));
  if (FT1 != NULL)
    CHECK(hipMemcpy(gFT1, FT1, 1024, hipMemcpyHostToDevice));
  if (FT2 != NULL)
    CHECK(hipMemcpy(gFT2, FT2, 1024, hipMemcpyHostToDevice));
  if (FT3 != NULL)
    CHECK(hipMemcpy(gFT3, FT3, 1024, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gFSb, FSb, 1024, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gaes_edrk, aes_edrk, 256, hipMemcpyHostToDevice));
  if (IV != NULL)
    CHECK(hipMemcpy(gIV, IV, 12, hipMemcpyHostToDevice));
  if (IV != NULL)
    CHECK(hipMemset(gIV+3,0,(size_t)4));
  CHECK(hipMemset(gout,0,(size_t)SIZE));
  t_[tc++] = wallclock();
  g.y = g.z = 1;
  g.x = (n_aes_block+255)/256;
  g.x *= tbp;
  while (g.x >= 65536) {
    g.x /= 2;
    g.y *= 2;
    while ((g.x*g.y)<((n_aes_block+255)/256)*tbp)
      g.x++;
  }
  b.y = b.z = 1;
  b.x = 256;
  printf("%d/%d,%d\n", g.x,g.y,b.x);
  t_[tc++] = wallclock();
  /* start the AES-in-nearly-CTR mode straight away */
  fct<<<g,b>>>(gin,gout,gaes_edrk,n_aes_block, gFT0, gFT1, gFT2, gFT3, gFSb, gIV);
  {
    t_[tc++] = wallclock();
    memset(temp,0,16);
    aes_encrypt(temp,(uint32_t*)H,aes_edrk);
    memcpy(temp, IV, 12);
    ((unsigned char*)temp)[15] = 1;
    aes_encrypt(temp,(uint32_t*)T,aes_edrk);
    t_[tc++] = wallclock();
  }
  CHECK(hipDeviceSynchronize());
  t_[tc++] = wallclock();
  CHECK(hipMemcpy(out3,gout,SIZE,hipMemcpyDeviceToHost)); 
  t_[tc++] = wallclock();
  {
    memset(accum,0,16);
    for (i = 0 ; i < SIZE ; i+=16) {
      addmul(accum,out3+i,16,H);
    }
    t_[tc++] = wallclock();
    unsigned char fb[16];
    memset(fb,0,16);
    (*(unsigned long long*)&fb[0]) = _bswap64((unsigned long long)(8*0));
    (*(unsigned long long*)&fb[8]) = _bswap64((unsigned long long)(8*SIZE));
    addmul(accum,fb,16,H);
    for (i = 0;i < 16;++i)
      (out3+SIZE)[i] = T[i] ^ accum[i];
    t_[tc++] = wallclock();
  }
  CHECK(hipFree(gin));
  CHECK(hipFree(gout));
  if (FT0 != NULL)
    CHECK(hipFree(gFT0));
  if (FT1 != NULL)
    CHECK(hipFree(gFT1));
  if (FT2 != NULL)
    CHECK(hipFree(gFT2));
  if (FT3 != NULL)
    CHECK(hipFree(gFT3));
  CHECK(hipFree(gFSb));
  CHECK(hipFree(gaes_edrk));
  if (IV != NULL)
    CHECK(hipFree(gIV));
  t_[tc++] = wallclock();
  count = 0;
  for (i = 0 ; i < SIZE+16 && count<10 ; i++) {
    if (out2[i] != out3[i]) {
      fprintf(stderr, "out2[%" PRIu64 "] != out3[%" PRIu64 "] : %02x != %02x\n", i, i, out2[i], out3[i]);
      count++;
    }
  }
  if (count)
    exit(-1);
  printf("Total = %lf\n", t_[tc-1]-t_[0]);
  for (int k = 1 ; k < tc ; k++) {
    printf("  t_[%d] - t_[%d] = %lf\n", k, k-1, t_[k]-t_[k-1]);
  }
  // 6 was the compute call
  t1 = t_[6];
  t0 = t_[4];
  printf("%lf AES: Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
  // 9 was the compute call
  t1 = t_[9];
  t0 = t_[7];
  printf("%lf GCM Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
  t1 = t_[tc-2];
  t0 = t_[1];
  printf("%lf NOMALLOCFREE Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
  t1 = t_[tc-1];
  t0 = t_[0];
  printf("%lf Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
    
}

void test_cuda_cpy(const uint8_t *in, uint8_t *out3, const uint8_t* out2, const uint32_t *aes_edrk,
               const uint32_t *FT0, const uint32_t *FT1, const uint32_t *FT2, const uint32_t *FT3,
               const uint32_t* IV, const uint16_t* FSbSq,
               aes_encrypt_cuda_proto fct, const int tbp) {
  uint64_t i, j;
  double t_[10], t0, t1;
  int count;
    int tc = 0;
    dim3 g, b;
    uint32_t *gin, *gout;
    uint32_t *gFT0 = NULL, *gFT1 = NULL, *gFT2 = NULL, *gFT3 = NULL;
    uint32_t *gFSb;
    uint32_t *gaes_edrk;
    uint32_t *gIV = NULL;
    uint16_t * gFSbSq;
    uint32_t n_aes_block = (SIZE+15)/16;
    hipError_t err;
    hipArray *FSbSqArray;
    hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

    t_[tc++] = wallclock();
    CHECK(hipMalloc((void**)&gin,(size_t)SIZE));
    CHECK(hipMalloc((void**)&gout,(size_t)SIZE));
    if (FT0 != NULL)
      CHECK(hipMalloc((void**)&gFT0,(size_t)1024));
    if (FT1 != NULL)
      CHECK(hipMalloc((void**)&gFT1,(size_t)1024));
    if (FT2 != NULL)
      CHECK(hipMalloc((void**)&gFT2,(size_t)1024));
    if (FT3 != NULL)
      CHECK(hipMalloc((void**)&gFT3,(size_t)1024));
    CHECK(hipMalloc((void**)&gFSb,(size_t)1024));
    CHECK(hipMalloc((void**)&gaes_edrk,(size_t)256));
    if (IV != NULL)
      CHECK(hipMalloc((void**)&gIV,(size_t)16));
    t_[tc++] = wallclock();
    if (FSbSq != NULL) {
//       CHECK(hipMallocArray(&FSbSqArray,&channelDesc,65536,0,hipArrayDefault));
      CHECK(hipMalloc(&gFSbSq,131072));
    }
    CHECK(hipMemcpy(gin, in, SIZE, hipMemcpyHostToDevice));
    if (FT0 != NULL)
      CHECK(hipMemcpy(gFT0, FT0, 1024, hipMemcpyHostToDevice));
    if (FT1 != NULL)
      CHECK(hipMemcpy(gFT1, FT1, 1024, hipMemcpyHostToDevice));
    if (FT2 != NULL)
      CHECK(hipMemcpy(gFT2, FT2, 1024, hipMemcpyHostToDevice));
    if (FT3 != NULL)
      CHECK(hipMemcpy(gFT3, FT3, 1024, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gFSb, FSb, 1024, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gaes_edrk, aes_edrk, 256, hipMemcpyHostToDevice));
    if (IV != NULL)
      CHECK(hipMemcpy(gIV, IV, 16, hipMemcpyHostToDevice));
    if (FSbSq != NULL) {
//       CHECK(hipMemcpyToArray(FSbSqArray,
//                               0,
//                               0,
//                               FSbSq,
//                               131072,
//                               hipMemcpyHostToDevice));
//       tFSbSq.addressMode[0] = hipAddressModeWrap;
//       tFSbSq.filterMode = hipFilterModePoint;
//       tFSbSq.normalized = false;
//       CHECK(hipBindTextureToArray(tFSbSq, FSbSqArray, channelDesc));
      CHECK(hipMemcpy(gFSbSq,FSbSq,131072,hipMemcpyHostToDevice));
      CHECK(hipBindTexture(NULL,tFSbSq,gFSbSq,channelDesc,131072));
    }
    CHECK(hipMemset(gout,0,(size_t)SIZE));
    t_[tc++] = wallclock();
    g.y = g.z = 1;
    g.x = (n_aes_block+255)/256;
    g.x *= tbp;
    while (g.x >= 65536) {
      g.x /= 2;
      g.y *= 2;
      while ((g.x*g.y)<((n_aes_block+255)/256)*tbp)
        g.x++;
    }
    b.y = b.z = 1;
    b.x = 256;
//     printf("%d/%d,%d\n", g.x,g.y,b.x);
    t_[tc++] = wallclock();
    fct<<<g,b>>>(gin,gout,gaes_edrk,n_aes_block, gFT0, gFT1, gFT2, gFT3, gFSb, gIV);
    CHECK(hipDeviceSynchronize());
    t_[tc++] = wallclock();
    CHECK(hipMemcpy(out3,gout,SIZE,hipMemcpyDeviceToHost)); 
    t_[tc++] = wallclock();
    CHECK(hipFree(gin));
    CHECK(hipFree(gout));
    if (FT0 != NULL)
      CHECK(hipFree(gFT0));
    if (FT1 != NULL)
      CHECK(hipFree(gFT1));
    if (FT2 != NULL)
      CHECK(hipFree(gFT2));
    if (FT3 != NULL)
      CHECK(hipFree(gFT3));
    CHECK(hipFree(gFSb));
    CHECK(hipFree(gaes_edrk));
    if (IV != NULL)
      CHECK(hipFree(gIV));
    if (FSbSq != NULL) {
//       CHECK(hipUnbindTexture(tFSbSq));
//       CHECK(hipFreeArray(FSbSqArray));
      CHECK(hipFree(gFSbSq));
    }
    t_[tc++] = wallclock();
    count = 0;
    for (i = 0 ; i < SIZE && count<10 ; i++) {
      if (out2[i] != out3[i]) {
        fprintf(stderr, "out2[%" PRIu64 "] != out3[%" PRIu64 "] : %02x != %02x\n", i, i, out2[i], out3[i]);
        count++;
      }
    }
    if (count)
      exit(-1);
    printf("Total = %lf\n", t_[tc-1]-t_[0]);
    for (int k = 1 ; k < tc ; k++) {
      printf("  t_[%d] - t_[%d] = %lf\n", k, k-1, t_[k]-t_[k-1]);
    }
    // 4 was the compute call
    t1 = t_[4];
    t0 = t_[3];
    fflush(stderr);
    printf("%lf Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
sleep(1);
}



void test_cuda_nocpy(const uint8_t *in, uint8_t *out3, const uint8_t* out2, const uint32_t *aes_edrk,
               const uint32_t *FT0, const uint32_t *FT1, const uint32_t *FT2, const uint32_t *FT3,
               const uint32_t* IV, const uint16_t* FSbSq,
               aes_encrypt_cuda_proto fct, const int tbp) {
  uint64_t i, j;
  double t_[10], t0, t1;
  int count;
    int tc = 0;
    dim3 g, b;
    uint32_t *gin, *gout;
    uint32_t *gFT0 = NULL, *gFT1 = NULL, *gFT2 = NULL, *gFT3 = NULL;
    uint32_t *gFSb;
    uint32_t *gaes_edrk;
    uint32_t *gIV = NULL;
    uint16_t * gFSbSq;
    uint32_t n_aes_block = (SIZE+15)/16;
    hipError_t err;
    hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

    t_[tc++] = wallclock();
    CHECK(hipHostGetDevicePointer((void**)&gin,(void*)in,0));
    CHECK(hipHostGetDevicePointer((void**)&gout,(void*)out3,0));
    if (FT0 != NULL)
      CHECK(hipHostGetDevicePointer((void**)&gFT0,(void*)FT0,0));
    if (FT1 != NULL)
      CHECK(hipHostGetDevicePointer((void**)&gFT1,(void*)FT1,0));
    if (FT2 != NULL)
      CHECK(hipHostGetDevicePointer((void**)&gFT2,(void*)FT2,0));
    if (FT3 != NULL)
      CHECK(hipHostGetDevicePointer((void**)&gFT3,(void*)FT3,0));
    CHECK(hipHostGetDevicePointer((void**)&gFSb,(void*)FSb,0));
    CHECK(hipHostGetDevicePointer((void**)&gaes_edrk,(void*)aes_edrk,0));
    if (IV != NULL)
      CHECK(hipHostGetDevicePointer((void**)&gIV,(void*)IV,0));
    t_[tc++] = wallclock();
    if (FSbSq != NULL) {
      fprintf(stderr, "FSbSq != NULL && nocpy incompatible\n");
      return;
    }
//     CHECK(hipMemset(gout,0,(size_t)SIZE));
    t_[tc++] = wallclock();
    g.y = g.z = 1;
    g.x = (n_aes_block+255)/256;
    g.x *= tbp;
    while (g.x >= 65536) {
      g.x /= 2;
      g.y *= 2;
      while ((g.x*g.y)<((n_aes_block+255)/256)*tbp)
        g.x++;
    }
    b.y = b.z = 1;
    b.x = 256;
//     printf("%d/%d,%d\n", g.x,g.y,b.x);
    t_[tc++] = wallclock();
    fct<<<g,b>>>(gin,gout,gaes_edrk,n_aes_block, gFT0, gFT1, gFT2, gFT3, gFSb, gIV);
    CHECK(hipDeviceSynchronize());
    t_[tc++] = wallclock();
    t_[tc++] = wallclock();
    t_[tc++] = wallclock();
    count = 0;
    for (i = 0 ; i < SIZE && count<10 ; i++) {
      if (out2[i] != out3[i]) {
        fprintf(stderr, "out2[%" PRIu64 "] != out3[%" PRIu64 "] : %02x != %02x\n", i, i, out2[i], out3[i]);
        count++;
      }
    }
    if (count)
      exit(-1);
    printf("Total = %lf\n", t_[tc-1]-t_[0]);
    for (int k = 1 ; k < tc ; k++) {
      printf("  t_[%d] - t_[%d] = %lf\n", k, k-1, t_[k]-t_[k-1]);
    }
    // 4 was the compute call
    t1 = t_[4];
    t0 = t_[3];
    fflush(stderr);
    printf("%lf Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
sleep(1); 
}


#ifdef NOCOPY
#define test_cuda test_cuda_nocpy
uint32_t* FT0;
uint32_t* FT1;
uint32_t* FT2;
uint32_t* FT3;
uint32_t* FSb;
#else
#define test_cuda test_cuda_cpy
#endif


int main(int argc, char **argv) {
  uint32_t keyr[8] = {0xc47b0294, 0xdbbbee0f, 0xec4757f2, 0x2ffeee35, 0x87ca4730, 0xc3d33b69, 0x1df38bab, 0x076bc558 };/* 46f2fb34 2d6f0ab4 77476fc50 1242c5f on 0[128] */
  uint32_t key[8];
  uint64_t i, j;
  double t0, t1;
  int count;
  hipError_t err;
  uint16_t FSbSq[65536];
  uint8_t *out1 = new uint8_t[SIZE+16]; /* +16 -> AES-GCM tag */
  uint8_t *out2 = new uint8_t[SIZE+16];

#ifdef NOCOPY
  uint8_t *in;
  uint8_t *out3;
  uint32_t *IV;
  uint32_t *aes_edrk;
  uint32_t *aes_edrk_diag;

  CHECK(hipSetDeviceFlags(hipDeviceMapHost));

  CHECK(hipHostAlloc(&in,SIZE,hipHostMallocMapped));
  CHECK(hipHostAlloc(&out3,SIZE+16,hipHostMallocMapped));
  CHECK(hipHostAlloc(&IV,16,hipHostMallocMapped));
  CHECK(hipHostAlloc(&aes_edrk,256,hipHostMallocMapped));
  CHECK(hipHostAlloc(&aes_edrk_diag,256,hipHostMallocMapped));
  CHECK(hipHostAlloc(&FT0,1024,hipHostMallocMapped));
  CHECK(hipHostAlloc(&FT1,1024,hipHostMallocMapped));
  CHECK(hipHostAlloc(&FT2,1024,hipHostMallocMapped));
  CHECK(hipHostAlloc(&FT3,1024,hipHostMallocMapped));
  CHECK(hipHostAlloc(&FSb,1024,hipHostMallocMapped));
  memcpy(FT0,FT0_,1024);
  memcpy(FT1,FT1_,1024);
  memcpy(FT2,FT2_,1024);
  memcpy(FT3,FT3_,1024);
  memcpy(FSb,FSb_,1024);
#else
  uint8_t *in = new uint8_t[SIZE];
  uint8_t *out3 = new uint8_t[SIZE+16];
  uint32_t IV[4];
  uint32_t aes_edrk[64];
  uint32_t aes_edrk_diag[64];
#endif

#if 0 //def NOCOPY
  CHECK(hipHostRegister(in,SIZE,hipHostRegisterMapped));
  CHECK(hipHostRegister(out3,SIZE+16,hipHostRegisterMapped));
  CHECK(hipHostRegister(IV,16,hipHostRegisterMapped));
  CHECK(hipHostRegister(aes_edrk,256,hipHostRegisterMapped));
  CHECK(hipHostRegister(aes_edrk_diag,256,hipHostRegisterMapped));
  CHECK(hipHostRegister(FT0,1024,hipHostRegisterMapped));
  CHECK(hipHostRegister(FT1,1024,hipHostRegisterMapped));
  CHECK(hipHostRegister(FT2,1024,hipHostRegisterMapped));
  CHECK(hipHostRegister(FT3,1024,hipHostRegisterMapped));
  CHECK(hipHostRegister(FSb,1024,hipHostRegisterMapped));
#endif
  
  for (i = 0 ; i < 256 ; i++) {
    for (j = 0 ; j < 256 ; j++) {
      FSbSq[j+i*256] = FSb[j] | (FSb[i]<<8);
    }
  }
  
  for (i = 0 ; i < 64 ; i++)
    aes_edrk[i] = 0;

  srandom(0); // reproducible

  for (i = 0 ; i < 4 ; i++)
    IV[i] = random();

  for (i = 0 ; i < SIZE/4 ; i++)
    ((uint32_t*)in)[i] = random();
  
  for (i = 0 ; i < SIZE ; i++)
    out1[i] = out2[i] = out3[i] = 0;

  for (i = 0 ; i < 8 ; i++) {
    key[i] = __builtin_bswap32(keyr[i]);
  }
  aes_set_key(key, aes_edrk);
  /* ** diagonalization of subkeys */
  /* first four are not diagonalized */
  for (i = 0 ; i < 4 ; i++) {
    aes_edrk_diag[i] = aes_edrk[i];
  }
  /* then all but last four are */
  for (i = 4 ; i < 56 ; i+= 4) {
    diag1cpu(aes_edrk_diag+i, aes_edrk+i);
  }
  /* last four */
  for (i = 56 ; i < 64 ; i++) {
    aes_edrk_diag[i] = aes_edrk[i];
  }
  
#define TEST_CUDA(K,FT0,FT1,FT2,FT3,IV,FSQ,FN,TBP)      \
  printf("Testing "#FN" ...\n");fflush(stdout);         \
  test_cuda(in, out3, out2, K,                          \
            FT0, FT1, FT2, FT3, IV,FSQ,                 \
            &FN, TBP); fflush(stderr);fflush(stdout);
  
#ifdef TEST_ECB
#if 0
  t0 = wallclock();
  for (i = 0 ; i < SIZE ; i+= 16) {
    aes_encrypt((uint32_t*)(in+i), (uint32_t*)(out1+i), aes_edrk);
  }
  t1 = wallclock();
  printf("%lf Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
//   for (i = 0 ; i < SIZE ; i+= 16) {
//     print16c((out1+i));
//   }


  for (i = 0 ; i < SIZE ; i+= 16) {
    aes_decrypt((uint32_t*)(out1+i), (uint32_t*)(in+i), aes_edrk);
  }
  count=0;
  for (i = 0 ; i < SIZE && count<10 ; i++) {
    if (in[i] != (i&0xFF)) {
      fprintf(stderr, "in[%" PRIu64 "] = 0x%02x != 0x%02x\n", i, in[i], i&0xFF);
      count++;
    }
  }
  if (count)
    exit(-1);
#endif

  {
    CryptoPP::ECB_Mode<CryptoPP::AES>::Encryption e((unsigned char*)key, 32);
    t0 = wallclock();
    e.ProcessData((unsigned char*)out2, (unsigned char*)in, SIZE);
    t1 = wallclock();
    printf("%lf Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
    //   for (i = 0 ; i < SIZE ; i+= 16) {
    //     print16c((out2+i));
    //   }
  }

#if 0
  count = 0;
  for (i = 0 ; i < SIZE && count<10 ; i++) {
    if (out1[i] != out2[i]) {
      fprintf(stderr, "out1[%" PRIu64 "] != out2[%" PRIu64 "] : %02x != %02x\n", i, i, out1[i], out2[i]);
      count++;
    }
  }
  if (count)
    exit(-1);
#endif

/* start things */
  TEST_CUDA(aes_edrk, FT0, NULL, NULL, NULL, NULL, NULL, aes_encrypt_cuda_FT_SEQ1_PRMT_32nocoalnocoal, 1);

#include "aes_cuda_ecb.h"

  TEST_CUDA(aes_edrk, FT0, NULL, NULL, NULL, NULL, NULL, aes_encrypt_cuda_half, 2);
  TEST_CUDA(aes_edrk, FT0, NULL, NULL, NULL, NULL, NULL, aes_encrypt_cuda_quarter, 4);
#endif

#ifdef TEST_CTR
  {
    CryptoPP::CTR_Mode<CryptoPP::AES>::Encryption s;
    s.SetKeyWithIV((unsigned char*)key, 32, (unsigned char*)IV);
    t0 = wallclock();
    s.ProcessString((unsigned char*)out2, (unsigned char*)in, SIZE);
    t1 = wallclock();
    printf("%lf Mbytes/seconds (%lf in %lf)\n", ((double)SIZE/(t1-t0))/1000000., (double)SIZE/1000000., t1-t0);
  }


/* start things */
  TEST_CUDA(aes_edrk, FT0, NULL, NULL, NULL, IV, NULL, aes_ctr_cuda_FT_SEQ1_PRMT_32nocoalnocoal, 1);

  #include "aes_cuda_ctr.h"

//   TEST_CUDA(aes_edrk, FT0, NULL, NULL, NULL, IV, NULL, aes_ctr_cuda_half, 2);
//   TEST_CUDA(aes_edrk, FT0, NULL, NULL, NULL, IV, NULL, aes_ctr_cuda_quarter, 4);
#endif

#ifdef TEST_GCM
  try {
    double t2;
    std::string cipher;
    CryptoPP::GCM<CryptoPP::AES>::Encryption e;
    e.SetKeyWithIV((unsigned char*)key, 32, (unsigned char*)IV, 12);
    CryptoPP::AuthenticatedEncryptionFilter aef(e, new CryptoPP::StringSink( cipher ), false, 16);
//     aef.ChannelPut(CryptoPP::AAD_CHANNEL, ad, adlen);
//     aef.ChannelMessageEnd(CryptoPP::AAD_CHANNEL);
    t0 = wallclock();
//     aef.ChannelPut(CryptoPP::AAD_CHANNEL, (byte*)NULL, (size_t)0);
//     aef.ChannelMessageEnd(CryptoPP::AAD_CHANNEL);
    aef.ChannelPut(CryptoPP::DEFAULT_CHANNEL, in, SIZE);
    aef.ChannelMessageEnd(CryptoPP::DEFAULT_CHANNEL);
    t1 = wallclock();
    memcpy(out2, cipher.c_str(), SIZE+16);
    t2 = wallclock();
    printf("%lf Mbytes/seconds (%lf in %lf) [%lf for %lf]\n",
           ((double)SIZE/(t1-t0))/1000000.,
           (double)SIZE/1000000., t1-t0,
           ((double)SIZE/(t2-t0))/1000000., t2-t0);
  } catch (CryptoPP::Exception& e ) {
    fprintf(stderr, "Oups, Crypto++ AES-GCM failed");
  }
  { 
    EVP_CIPHER_CTX x;
    int outlen = 0;
    int ok = 1;
    
    t0 = wallclock();
    EVP_CIPHER_CTX_init(&x);
    if (ok == 1) ok = EVP_EncryptInit_ex(&x,EVP_aes_256_gcm(),0,0,0);
    if (ok == 1) ok = EVP_CIPHER_CTX_ctrl(&x,EVP_CTRL_GCM_SET_IVLEN,12,0);
    if (ok == 1) ok = EVP_EncryptInit_ex(&x,0,0,(const unsigned char *)key,(const unsigned char *)IV);
//     if (ok == 1) ok = EVP_EncryptUpdate(&x,0,&outlen,ad,adlen);
    if (ok == 1) ok = EVP_EncryptUpdate(&x,out3,&outlen,in,SIZE);
    if (ok == 1) ok = EVP_EncryptFinal_ex(&x,out3,&outlen);
    if (ok == 1) ok = EVP_CIPHER_CTX_ctrl(&x,EVP_CTRL_GCM_GET_TAG,16,out3 + SIZE);
    EVP_CIPHER_CTX_cleanup(&x);
    t1 = wallclock();
    if (ok != 1)
      fprintf(stderr, "Oups, openssl AES-GCM failed");
    count = 0;
    for (i = 0 ; i < SIZE+16 && count<10 ; i++) {
      if (out2[i] != out3[i]) {
        fprintf(stderr, "out2[%" PRIu64 "] != out3[%" PRIu64 "] : %02x != %02x\n", i, i, out2[i], out3[i]);
        count++;
      }
    }
    if (count)
      exit(-1);
    printf("%lf Mbytes/seconds (%lf in %lf)\n",
           ((double)SIZE/(t1-t0))/1000000.,
           (double)SIZE/1000000., t1-t0);
  }
  {
    printf("Testing  aes_gcm_cuda_BTB320_PRMT_8nocoalnocoal ...\n");
    fflush(stdout);
    test_cuda_gcm(in, out3, out2, aes_edrk,
                  FT0, NULL, NULL, NULL,
                  IV, /* NULL, */
                  &aes_gcm_cuda_BTB320_PRMT_8nocoalnocoal, 1);
  }
#endif


#if 0 //def NOCOPY
  CHECK(hipHostUnregister(in));
  CHECK(hipHostUnregister(out3));
  CHECK(hipHostUnregister(IV));
  CHECK(hipHostUnregister(aes_edrk));
  CHECK(hipHostUnregister(aes_edrk_diag));
  CHECK(hipHostUnregister(FT0));
  CHECK(hipHostUnregister(FT1));
  CHECK(hipHostUnregister(FT2));
  CHECK(hipHostUnregister(FT3));
  CHECK(hipHostUnregister(FSb));
#endif

#ifdef NOCOPY
  CHECK(hipHostFree(in));
  CHECK(hipHostFree(out3));
  CHECK(hipHostFree(IV));
  CHECK(hipHostFree(aes_edrk));
  CHECK(hipHostFree(aes_edrk_diag));
  CHECK(hipHostFree(FT0));
  CHECK(hipHostFree(FT1));
  CHECK(hipHostFree(FT2));
  CHECK(hipHostFree(FT3));
  CHECK(hipHostFree(FSb));
#else  
  delete(in);
  delete(out3);
#endif
  delete(out1);
  delete(out2);

  return 0;
}
