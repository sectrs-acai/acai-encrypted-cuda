#include "hip/hip_runtime.h"
/*
Copyright (c) 2014, Romain Dolbeau, unless otherwise noted.
No claims is made upon work by others.

For the work by Romain Dolbeau:
All rights reserved

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of the <organization> nor the
      names of its contributors may be used to endorse or promote products
      derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

For the work by others:
See the indicated reference for the relevant license.
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#if __cplusplus >= 201103L
#include <cstdint>
#else
#define uint64_t unsigned long long
#define uint32_t unsigned int
#define uint16_t unsigned short
#define uint8_t unsigned char
#endif

#include <cryptopp/aes.h>
#include <cryptopp/modes.h>
#include <cryptopp/gcm.h>
#include <cryptopp/filters.h>
#include <cryptopp/channels.h>

#include <openssl/evp.h>

#include "aes_common.h"
#include "aes_scalar.h"
#include "aes_gcm.h"

texture<unsigned short, 1, hipReadModeElementType> tFSbSq;

#include "aes_gpu.h"
#include "gpu_aes_gcm.h"

/* AES256GCM encryption.
   this is from the supercop benchmark <http://bench.cr.yp.to/supercop.html>
   directory "supercop-$VERSION/crypto_aead/aes256gcmv1/openssl"
*/
int crypto_aead_encrypt_openssl(
  unsigned char *c,unsigned long long *clen,
  const unsigned char *m,unsigned long long mlen,
  const unsigned char *ad,unsigned long long adlen,
  const unsigned char *nsec,
  const unsigned char *npub,
  const unsigned char *k
)
{
  EVP_CIPHER_CTX x;
  int outlen = 0;
  int ok = 1;

  if (adlen > 536870912) return -111;
  /* OpenSSL needs to put lengths into an int */
  if (mlen > 536870912) return -111;

  EVP_CIPHER_CTX_init(&x);
  if (ok == 1) ok = EVP_EncryptInit_ex(&x,EVP_aes_256_gcm(),0,0,0);
  if (ok == 1) ok = EVP_CIPHER_CTX_ctrl(&x,EVP_CTRL_GCM_SET_IVLEN,12,0);
  if (ok == 1) ok = EVP_EncryptInit_ex(&x,0,0,k,npub);
  if (ok == 1) ok = EVP_EncryptUpdate(&x,0,&outlen,ad,adlen);
  if (ok == 1) ok = EVP_EncryptUpdate(&x,c,&outlen,m,mlen);
  if (ok == 1) ok = EVP_EncryptFinal_ex(&x,c,&outlen);
  if (ok == 1) ok = EVP_CIPHER_CTX_ctrl(&x,EVP_CTRL_GCM_GET_TAG,16,c + mlen);
  EVP_CIPHER_CTX_cleanup(&x);

  if (ok == 1) {
    *clen = mlen + 16;
    return 0;
  }
  return -111;
}

/* AES256GCM decryption.
   this is from the supercop benchmark <http://bench.cr.yp.to/supercop.html>
   directory "supercop-$VERSION/crypto_aead/aes256gcmv1/openssl"
*/
int crypto_aead_decrypt_openssl(
  unsigned char *m,unsigned long long *mlen,
  unsigned char *nsec,
  const unsigned char *c,unsigned long long clen,
  const unsigned char *ad,unsigned long long adlen,
  const unsigned char *npub,
  const unsigned char *k
)
{
  EVP_CIPHER_CTX x;
  int outlen = 0;
  int ok = 1;
  
  if (adlen > 536870912) return -111; 
  /* OpenSSL needs to put lengths into an int */
  if (clen > 536870912) return -111;

  if (clen < 16) return -1;
  clen -= 16;

  EVP_CIPHER_CTX_init(&x);
  if (ok == 1) ok = EVP_DecryptInit_ex(&x,EVP_aes_256_gcm(),0,0,0);
  if (ok == 1) ok = EVP_CIPHER_CTX_ctrl(&x,EVP_CTRL_GCM_SET_IVLEN,12,0);
  if (ok == 1) ok = EVP_CIPHER_CTX_ctrl(&x,EVP_CTRL_GCM_SET_TAG,16,(unsigned char *) c + clen);
  if (ok == 1) ok = EVP_DecryptInit_ex(&x,0,0,k,npub);
  if (ok == 1) ok = EVP_DecryptUpdate(&x,0,&outlen,ad,adlen);
  if (ok == 1) ok = EVP_DecryptUpdate(&x,m,&outlen,c,clen);
  if (ok == 1) ok = EVP_DecryptFinal_ex(&x,m + clen,&outlen);
  EVP_CIPHER_CTX_cleanup(&x);

  if (ok == 1) {
    *mlen = clen;
    return 0;
  }
  return -1; /* forgery; XXX: or out of memory? hmmm */
}

/* AES256GCM encryption.
   this is from the supercop benchmark <http://bench.cr.yp.to/supercop.html>
   directory "supercop-$VERSION/crypto_aead/aes256gcmv1/cryptopp"
*/
int crypto_aead_encrypt_cryptopp(
  unsigned char *c,unsigned long long *clen,
  const unsigned char *m,unsigned long long mlen,
  const unsigned char *ad,unsigned long long adlen,
  const unsigned char *nsec,
  const unsigned char *npub,
  const unsigned char *k
)
{
  try {
    std::string cipher;
    CryptoPP::GCM<CryptoPP::AES>::Encryption e;
    e.SetKeyWithIV(k, 32, npub, 12);
    CryptoPP::AuthenticatedEncryptionFilter aef(e, new CryptoPP::StringSink( cipher ), false, 16);
    aef.ChannelPut(CryptoPP::AAD_CHANNEL, ad, adlen);
    aef.ChannelMessageEnd(CryptoPP::AAD_CHANNEL);
    aef.ChannelPut(CryptoPP::DEFAULT_CHANNEL, m, mlen);
    aef.ChannelMessageEnd(CryptoPP::DEFAULT_CHANNEL);
    
    *clen = mlen + 16;
    
    memcpy(c, cipher.c_str(), *clen);
    
    return 0;
  } catch (CryptoPP::Exception& e ) {
    return -111;
  }
}

/* AES256GCM decryption.
   this is from the supercop benchmark <http://bench.cr.yp.to/supercop.html>
   directory "supercop-$VERSION/crypto_aead/aes256gcmv1/cryptopp"
*/
int crypto_aead_decrypt_cryptopp(
  unsigned char *m,unsigned long long *outputmlen,
  unsigned char *nsec,
  const unsigned char *c,unsigned long long clen,
  const unsigned char *ad,unsigned long long adlen,
  const unsigned char *npub,
  const unsigned char *k
)
{
  try {
    std::string plain;
    CryptoPP::GCM<CryptoPP::AES>::Decryption d;
    d.SetKeyWithIV(k, 32, npub, 12);
    CryptoPP::AuthenticatedDecryptionFilter adf( d, NULL, CryptoPP::AuthenticatedDecryptionFilter::MAC_AT_BEGIN | CryptoPP::AuthenticatedDecryptionFilter::THROW_EXCEPTION, 16 );
    *outputmlen = clen-16;
    adf.ChannelPut(CryptoPP::DEFAULT_CHANNEL, c+clen-16, 16);
    adf.ChannelPut(CryptoPP::AAD_CHANNEL, ad, adlen); 
    adf.ChannelPut(CryptoPP::DEFAULT_CHANNEL, c, clen-16);
    adf.ChannelMessageEnd(CryptoPP::AAD_CHANNEL);
    adf.ChannelMessageEnd(CryptoPP::DEFAULT_CHANNEL);
    if (!adf.GetLastResult())
      return -111;
    adf.SetRetrievalChannel(CryptoPP::DEFAULT_CHANNEL);
    adf.Get(m, *outputmlen);
    return 0;
  } catch (CryptoPP::Exception& e ) {
    return -111;
  }
}

#define CHECK(K)                                                        \
  do {                                                                  \
    err = K;                                                            \
    if (err) {                                                          \
      fprintf(stderr, "Oups, "#K" failed with %d (%s)\n", err, hipGetErrorString(err)); \
      fflush(stderr);                                                   \
      exit(-2);                                                         \
    } } while (0)
#define CHECKRETRY46(K)                                                 \
  do {                                                                  \
    int tcount = 0;                                                     \
    do { err = K; tcount++; } while (err == 46 && tcount < 3);          \
    if (err) {                                                          \
      fprintf(stderr, "Oups, "#K" failed with %d (%s)\n", err, hipGetErrorString(err)); \
      fflush(stderr);                                                   \
      exit(-2);                                                         \
    } } while (0)

/* gpu pointers */
uint32_t *gin, *gout;
uint32_t *gFT0, *gFT1, *gFT2, *gFT3;
uint32_t *gFSb;
uint32_t *gaes_edrk;
uint32_t *gIV;
hipStream_t streams[2];

static inline void print16c(const uint8_t* buf) {
  uint64_t i;
  for(i = 0 ; i < 16 ; i++) {
    printf("%02x ", buf[i]);
    if (i%4==3)
      printf(" ");
  }
  printf("\n");
}

#ifndef GCM_CUDA_CHUNK_SIZE
#define GCM_CUDA_CHUNK_SIZE (2*1024*1024) /* must be a multiple of 4096 */
#endif
#ifndef GCM_CUDA_CHUNK_NUMBLOCK
#define GCM_CUDA_CHUNK_NUMBLOCK 8
#endif
//#define GCM_CUDA_ENCRYPT_BY_CHUNK /* whether we want to encrypt by chunk, i.e., overlap AES & GCM during encryption */
//#define GCM_CUDA_ENCRYPT_BY_CHUNK_FIXED_CHUNK /* whether the crypt size is fixed, and not size/GCM_CUDA_CHUNK_NUMBLOCK */
//#define GPU_NOXOR /* if defined, GPU doesn't do XOR - the CPU does */

/* initialization for the CUDA stuff.
   allocates GPU-side buffers, streams, ...
   can also set the cache config or pick
   a GPU.
   THE HYBRID CODE IS NOT REENTRANT !
   Don't use in more than one thread.
*/
static size_t gcm_cuda_max_size;
static int gcm_cuda_max_blocks;
void init_crypto_aead_cuda(const size_t SIZE, const int maxb) {
  hipError_t err;
  gcm_cuda_max_size = SIZE;
  gcm_cuda_max_blocks = maxb;
  CHECKRETRY46(hipSetDeviceFlags(hipDeviceScheduleYield));
  CHECKRETRY46(hipMalloc((void**)&gin,SIZE));
  CHECK(hipMalloc((void**)&gout,SIZE));
  CHECK(hipMalloc((void**)&gFT0,(size_t)1024));
  CHECK(hipMalloc((void**)&gFT1,(size_t)1024));
  CHECK(hipMalloc((void**)&gFT2,(size_t)1024));
  CHECK(hipMalloc((void**)&gFT3,(size_t)1024));
  CHECK(hipMalloc((void**)&gFSb,(size_t)1024));
  CHECK(hipMalloc((void**)&gaes_edrk,(size_t)256));
  CHECK(hipMalloc((void**)&gIV,(size_t)16*maxb));
  CHECK(hipMemcpy(gFT0, FT0, 1024, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gFT1, FT1, 1024, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gFT2, FT2, 1024, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gFT3, FT3, 1024, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(gFSb, FSb, 1024, hipMemcpyHostToDevice));
  int pmin, pmax;
  CHECK(hipDeviceGetStreamPriorityRange(&pmin,&pmax));
  /* Beware: priority is not actually suported on Jetson TK1
     (pmin == pmax == 0), only on Tesla/Quadro CC3.5+
  */
  /* minimum priority stream : AES */
  CHECK(hipStreamCreateWithPriority(&streams[0], hipStreamDefault, pmin));
  /* maximum priority stream : recover data from partial AES for pipelining */
  CHECK(hipStreamCreateWithPriority(&streams[1],hipStreamDefault, pmax));
  /*
  CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&aes_gcm_cuda_BTB320_PRMT_8nocoalnocoal), hipFuncCachePreferL1));
  CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&aes_gcm_cuda_BTB320_PRMT_8coalnocoal), hipFuncCachePreferShared));
  CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&aes_gcm_cuda_BTB320_PRMT_8nocoalcoal), hipFuncCachePreferShared));
  CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&aes_gcm_cuda_BTB320_PRMT_8coalcoal), hipFuncCachePreferShared));
  CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&aes_gcm_cuda_BTB32DIAGKEY0_PRMT_8coalcoal), hipFuncCachePreferL1));
  CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&aes_gcm_cuda_BTB32DIAGKEY0_PRMT_8nocoalnocoal), hipFuncCachePreferL1));
  */
}

/* free the resources allocated by the init function */
void finish_crypto_aead_cuda(void) {
  hipError_t err;
  CHECK(hipFree(gin));
  CHECK(hipFree(gout));
  CHECK(hipFree(gFT0));
  CHECK(hipFree(gFT1));
  CHECK(hipFree(gFT2));
  CHECK(hipFree(gFT3));
  CHECK(hipFree(gFSb));
  CHECK(hipFree(gaes_edrk));
  CHECK(hipFree(gIV));
  CHECK(hipStreamDestroy(streams[0]));
}

/* same interface as the supercop crypto_aead_encrypt()
   functions.
   *But* it requires initialization and clean-up
   */
int crypto_aead_encrypt_cuda(
  unsigned char *c,unsigned long long *clen,
  const unsigned char *m,unsigned long long mlen,
  const unsigned char *ad,unsigned long long adlen,
  const unsigned char *nsec,
  const unsigned char *npub,
  const unsigned char *k
)
{
  unsigned long long nfullgpuaesblock = mlen/(256*16);
  unsigned long long nfullaesblock;
  unsigned long long dataleft;
  uint32_t aes_edrk[64];
  uint32_t aes_edrk_diag[64];
  unsigned char accum[16], H[16], T[16], temp[16], fb[16];
  hipError_t err;
  //aes_gcm_cuda_BTB320_PRMT_8nocoalnocoal
  //aes_gcm_cuda_BTB32DIAGKEY0_PRMT_8coalcoal
#ifdef GPU_NOXOR
  aes_encrypt_cuda_proto fct = &aes_gcmnoxor_cuda_BTB32DIAGKEY0_PRMT_8coalcoal;
#else
  aes_encrypt_cuda_proto fct = &aes_gcm_cuda_BTB32DIAGKEY0_PRMT_8coalcoal;
#endif
#ifdef GCM_CUDA_ENCRYPT_BY_CHUNK
  unsigned long long nchunk;
  size_t gcm_cuda_chunk_size = GCM_CUDA_CHUNK_SIZE;
#ifndef GCM_CUDA_ENCRYPT_BY_CHUNK_FIXED_CHUNK
  gcm_cuda_chunk_size = ((mlen/GCM_CUDA_CHUNK_NUMBLOCK)+4095) & ~4095;
  if (gcm_cuda_chunk_size < (512*1024))
    gcm_cuda_chunk_size = 512*1024;
  if (gcm_cuda_chunk_size > 16*1024*1024)
    gcm_cuda_chunk_size = 16*1024*1024;
  /* fixme: check with gcm_cuda_max_blocks */
#endif
#endif
  int i, j;
  dim3 g, b;
  aes_set_key((const unsigned int*)k, aes_edrk);
  {
    /* ** diagonalization of subkeys */
    /* first four are not diagonalized */
    for (i = 0 ; i < 4 ; i++) {
      aes_edrk_diag[i] = aes_edrk[i];
    }
    /* then all but last four are */
    for (i = 4 ; i < 56 ; i+= 4) {
      diag1cpu(aes_edrk_diag+i, aes_edrk+i);
    }
    /* last four */
    for (i = 56 ; i < 64 ; i++) {
      aes_edrk_diag[i] = aes_edrk[i];
    }
  }
  *clen = mlen+16;
  if ((nfullgpuaesblock*256*16) > gcm_cuda_max_size) {
    nfullgpuaesblock = gcm_cuda_max_size/(256*16);
  }
  g.y = g.z = 1;
  g.x = nfullgpuaesblock;
  g.x *= 1; /* tbp */
  while (g.x >= 65536) {
    g.x /= 2;
    g.y *= 2;
  }
  nfullgpuaesblock = g.x*g.y/1; /* tbp */
  nfullaesblock = 256*nfullgpuaesblock;
#ifdef GCM_CUDA_ENCRYPT_BY_CHUNK
  nchunk = (nfullgpuaesblock+(gcm_cuda_chunk_size/4096)-1)/(gcm_cuda_chunk_size/4096);
  hipEvent_t event[nchunk];
#endif
  dataleft = mlen-(nfullgpuaesblock*256*16);
  b.y = b.z = 1;
  b.x = 256;
  if (nfullgpuaesblock>0) {
    memset(temp,0,16);
    memcpy(temp,npub,12);
    /* Since we're in non-pinned memory, hipMemcpyAsync probably won't help */
    //   CHECK(hipMemcpyAsync(gaes_edrk,aes_edrk,256,hipMemcpyHostToDevice,0));
    //   CHECK(hipMemcpyAsync(gIV,npub,12,hipMemcpyHostToDevice,0));
    //   CHECK(hipMemcpyAsync(gin,m,nfullgpuaesblock*256*16,hipMemcpyHostToDevice,0/*streams[0]*/));
    /* BEWARE which keys are send (diagonalized or not) */
    CHECK(hipMemcpy(gaes_edrk,aes_edrk_diag,256,hipMemcpyHostToDevice));
    //CHECK(hipMemcpy(gaes_edrk,aes_edrk,256,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gIV,temp,16,hipMemcpyHostToDevice));
#if !defined(GPU_NOXOR)
    CHECK(hipMemcpy(gin,m,nfullgpuaesblock*256*16,hipMemcpyHostToDevice));
#endif
#ifdef GCM_CUDA_ENCRYPT_BY_CHUNK
    if (nchunk > 1) {
      uint32_t bIV[(nchunk-1)*4];
      memset(bIV,0,(nchunk-1)*4*sizeof(uint32_t));
      for (i = 1 ; i < nchunk ; i++) {
        memcpy(bIV+(i-1)*4,npub,12);
        bIV[(4*i)-1] = __builtin_bswap32(i*gcm_cuda_chunk_size/16);
      }
      CHECK(hipMemcpy(gIV+4,bIV,(nchunk-1)*4*sizeof(uint32_t),hipMemcpyHostToDevice));
    }
    for (i = 0 ; i < nchunk ; i++) {
      CHECK(hipEventCreate(event+i));
    }
    for (i = 0 ; i < nchunk ; i++) {
      unsigned long long naesb = gcm_cuda_chunk_size/16;
      if (naesb > (nfullaesblock-(i*gcm_cuda_chunk_size/16)))
        naesb = nfullaesblock-(i*gcm_cuda_chunk_size/16);
      /* need to recompute grid for each chunk */
      g.y = g.z = 1;
      g.x = gcm_cuda_chunk_size/(256*16);
      g.x *= 1; /* tbp */
      /* problem if g.x >= 65535 ... won't happen (gcm_cuda_chunk_size <= 8*1024*1024 -> g.x <= 2048) */
      fct<<<g,b,0,streams[0]>>>(gin+gcm_cuda_chunk_size/sizeof(uint32_t)*i,
                                gout+gcm_cuda_chunk_size/sizeof(uint32_t)*i,
                                gaes_edrk,naesb,gFT0, gFT1, gFT2, gFT3, gFSb,
                                gIV+4*i);
      CHECK(hipEventRecord(event[i],streams[0]));
    }
#else
    fct<<<g,b,0,0/*streams[0]*/>>>(gin,gout,gaes_edrk,nfullaesblock, gFT0, gFT1, gFT2, gFT3, gFSb, gIV);
#endif // GCM_CUDA_ENCRYPT_BY_CHUNK
  }
  /* encrypt leftovers */
  memset(temp,0,16);
  memcpy(temp, npub, 12);
  for (i = nfullgpuaesblock*256 ; i < (mlen+15)/16 ; i++) {
    int max;
    ((uint32_t*)temp)[3] = __builtin_bswap32(i+2);
    aes_encrypt((uint32_t*)temp,(uint32_t*)H,aes_edrk);
    max = 16;
    if (mlen-i*16<16)
      max = mlen-i*16;
    for (j = 0 ; j < max; j++) {
      c[i*16+j]=m[i*16+j] ^ H[j];
    }
    dataleft -= max;
  } 
  assert(0 == dataleft);
  /* GCM on AD */
  memset(accum,0,16);
  memset(temp,0,16);
  memset(H,0,16);
  memset(T,0,16);
  memset(fb,0,16);
  aes_encrypt((uint32_t*)temp,(uint32_t*)H,aes_edrk);
  memcpy(temp, npub, 12);
  temp[15] = 1;
  aes_encrypt((uint32_t*)temp,(uint32_t*)T,aes_edrk);
  do_gcm(accum, H, ad, adlen);
#ifdef GCM_CUDA_ENCRYPT_BY_CHUNK
  if (nfullgpuaesblock > 0) {
    for (i = 0 ; i < nchunk ; i++) {
      unsigned long long naesb = gcm_cuda_chunk_size/16;
      if (naesb > (nfullaesblock-(i*gcm_cuda_chunk_size/16)))
        naesb = nfullaesblock-(i*gcm_cuda_chunk_size/16);
      CHECK(hipEventSynchronize(event[i]));
      CHECK(hipEventDestroy(event[i]));
      // Here we can't use hipMemcpy, as it run in the default stream,
      // and therefore will wait for all kernels execution before running
      // the first hipMemcpy.
      // however, using hipMemcpyAsync on streams[1] seems to work, despite
      // the fact the call will likely be synchronous.
      CHECK(hipMemcpyAsync(c+gcm_cuda_chunk_size*i,
                            gout+gcm_cuda_chunk_size/sizeof(uint32_t)*i,naesb*16,hipMemcpyDeviceToHost,streams[1]));
      CHECK(hipStreamSynchronize(streams[1]));
#if !defined(GPU_NOXOR)
      do_gcm(accum, H, c+gcm_cuda_chunk_size*i, naesb*16);
#else
      do_xor_gcm(accum, H, c+gcm_cuda_chunk_size*i, m+gcm_cuda_chunk_size*i, naesb*16);
#endif // !GPU_NOXOR
    }
  }
  do_gcm(accum, H, c+nfullaesblock*16, mlen-nfullaesblock*16);
#else
  if (nfullgpuaesblock > 0) {
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(c,gout,nfullgpuaesblock*256*16,hipMemcpyDeviceToHost));
  }
#if !defined(GPU_NOXOR)
  do_gcm(accum, H, c, mlen);
#else
  do_xor_gcm(accum, H, c, m, mlen);
#endif // !GPU_NOXOR
#endif // GCM_CUDA_ENCRYPT_BY_CHUNK
  (*(unsigned long long*)&fb[0]) = _bswap64((unsigned long long)(8*adlen));
  (*(unsigned long long*)&fb[8]) = _bswap64((unsigned long long)(8*mlen));
  addmul(accum,fb,16,H);
  for (i = 0;i < 16;++i)
    (c+mlen)[i] = T[i] ^ accum[i];
  return 0;
}

/* same interface as the supercop crypto_aead_decrypt()
   functions.
   *But* it requires initialization and clean-up
   */
int crypto_aead_decrypt_cuda(
  unsigned char *m,unsigned long long *outputmlen,
  unsigned char *nsec,
  const unsigned char *c,unsigned long long clen,
  const unsigned char *ad,unsigned long long adlen,
  const unsigned char *npub,
  const unsigned char *k
)
{
  unsigned long long nfullgpuaesblock =  (clen-16)/(256*16);
  unsigned long long nfullaesblock;
  unsigned long long dataleft;
  uint32_t aes_edrk[64];
  uint32_t aes_edrk_diag[64];
  unsigned char accum[16], H[16], T[16], temp[16], fb[16];
  hipError_t err;
  int i, j;
  int res = 0;
  dim3 g, b;
  aes_set_key((const unsigned int*)k, aes_edrk);
  {
    /* ** diagonalization of subkeys */
    /* first four are not diagonalized */
    for (i = 0 ; i < 4 ; i++) {
      aes_edrk_diag[i] = aes_edrk[i];
    }
    /* then all but last four are */
    for (i = 4 ; i < 56 ; i+= 4) {
      diag1cpu(aes_edrk_diag+i, aes_edrk+i);
    }
    /* last four */
    for (i = 56 ; i < 64 ; i++) {
      aes_edrk_diag[i] = aes_edrk[i];
    }
  }
  *outputmlen = clen-16;
  if ((nfullgpuaesblock*256*16) > gcm_cuda_max_size) {
    nfullgpuaesblock = gcm_cuda_max_size/(256*16);
  }
  g.y = g.z = 1;
  g.x = nfullgpuaesblock;
  g.x *= 1; /* tbp */
  while (g.x >= 65536) {
    g.x /= 2;
    g.y *= 2;
  }
  nfullgpuaesblock = g.x*g.y/1;
  nfullaesblock = 256 * nfullgpuaesblock;
  dataleft = (*outputmlen)-(nfullgpuaesblock*256*16);
  b.y = b.z = 1;
  b.x = 256;
  if (nfullgpuaesblock>0) {
  //   CHECK(hipMemcpyAsync(gaes_edrk,aes_edrk,256,hipMemcpyHostToDevice,0));
  //   CHECK(hipMemcpyAsync(gIV,npub,12,hipMemcpyHostToDevice,0));
  //   CHECK(hipMemcpyAsync(gin,m,nfullgpuaesblock*256*16,hipMemcpyHostToDevice,0/*streams[0]*/));
    /* BEWARE which keys are send (diagonalized or not) */
    CHECK(hipMemcpy(gaes_edrk,aes_edrk_diag,256,hipMemcpyHostToDevice));
    CHECK(hipMemset(gIV,0,16));
    CHECK(hipMemcpy(gIV,npub,12,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(gin,m,nfullgpuaesblock*256*16,hipMemcpyHostToDevice));
    aes_gcm_cuda_BTB32DIAGKEY0_PRMT_8coalcoal<<<g,b,0,0/*streams[0]*/>>>(gin,gout,gaes_edrk,nfullaesblock, gFT0, gFT1, gFT2, gFT3, gFSb, gIV);
  }
  /* decrypt leftovers */
  memset(temp,0,16);
  memcpy(temp, npub, 12);
  for (i = nfullgpuaesblock*256 ; i < ((*outputmlen)+15)/16 ; i++) {
    int max;
    ((uint32_t*)temp)[3] = __builtin_bswap32(i+2);
    aes_encrypt((uint32_t*)temp,(uint32_t*)H,aes_edrk);
    max = 16;
    if ((*outputmlen)-i*16<16)
      max = (*outputmlen)-i*16;
    for (j = 0 ; j < max; j++) {
      m[i*16+j]=c[i*16+j] ^ H[j];
    }
    dataleft -= max;
  } 
  assert(0 == dataleft);
  /* GCM on AD */
  memset(accum,0,16);
  memset(temp,0,16);
  memset(H,0,16);
  memset(T,0,16);
  memset(fb,0,16);
  aes_encrypt((uint32_t*)temp,(uint32_t*)H,aes_edrk);
  memcpy(temp, npub, 12);
  temp[15] = 1;
  aes_encrypt((uint32_t*)temp,(uint32_t*)T,aes_edrk);
  do_gcm(accum, H, ad, adlen);
  /* GCM on encrypted (async) */
  do_gcm(accum, H, c, *outputmlen);
  (*(unsigned long long*)&fb[0]) = _bswap64((unsigned long long)(8*adlen));
  (*(unsigned long long*)&fb[8]) = _bswap64((unsigned long long)(8*(*outputmlen)));
  addmul(accum,fb,16,H);
  for (i = 0;i < 16;++i)
    res |= T[i] ^ accum[i];
  if (nfullgpuaesblock > 0) {
    if (!res) {
      CHECK(hipDeviceSynchronize());
      CHECK(hipMemcpy(m,gout,nfullgpuaesblock*256*16,hipMemcpyDeviceToHost));
    } else {
      CHECK(hipDeviceSynchronize());
      CHECK(hipMemset(gout,0,nfullgpuaesblock*256*16));
      memset(m,0,(*outputmlen)); // we might have written up to 4095 bytes
    }
  }
  return res;
}
