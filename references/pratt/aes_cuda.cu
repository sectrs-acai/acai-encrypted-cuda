#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "aes_cuda.h"
#include "aes.h"
#include "table.h"




//copy block from inp to outp(1 block per thread)
__device__ void copy_block(uint8_t *inp, uint8_t *out, uint32_t offset){
  //word size traversal
  uint32_t *id = (uint32_t *)inp;
  uint32_t *od = (uint32_t *)out;
  for(int i = 0; i < 4; ++i){
    od[offset/4 + i] = id[offset/4 + i];
  }
}


//XOR round key with block(1 block per thread)
__device__ void add_round_key(uint8_t *block, uint8_t *key, uint32_t offset){
  //word size traversal
  uint32_t *b = (uint32_t *)block;
  uint32_t *k = (uint32_t *)key;
  for(int i = 0; i < 4; ++i){
    b[offset/4 + i] = b[offset/4 + i] ^ k[i];
  }  
}

//substitute block int sbox (1 block per thread)
__device__ void sub_bytes(uint8_t *block, uint32_t offset){
  for(int i = 0; i < 16; ++i){
    block[offset + i] = sbox[block[offset + i]];
  }
}


//mix columns by taking linear combinations in the field (1 block per thread)
__device__ void mix_columns(uint8_t *block, uint32_t offset){
  for(int i = 0; i < 4; ++i){ //iterate over columns
    uint8_t a[4];
    uint8_t b[4]; 
    uint8_t h;
  
    for(int j = 0; j < 4; ++j){
      a[j] = block[offset + 4*i + j];
      h = (uint8_t)((int8_t)a[j] >> 7);
      b[j] = a[j] << 1;
      b[j] ^= 0x1b & h;
    } 

    block[offset + 4*i + 0] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1];
    block[offset + 4*i + 1] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2];
    block[offset + 4*i + 2] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3];
    block[offset + 4*i + 3] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0]; 

  }
}


//shift rows left by 0,1,2,3 bytes respectively (1 block per thread)
__device__ void shift_rows(uint8_t *sblock, uint32_t offset){
  uint8_t tmp;

  uint8_t *block = sblock + offset; 

  //row 0 remains unshifted

  //shift row 1 left by 1
  tmp = block[1];
  block[1] = block[5];
  block[5] = block[9];
  block[9] = block[13];
  block[13] = tmp;

  //shift row 2 letf by 2
  tmp = block[2];
  block[2] = block[10];
  block[10] = tmp;

  tmp = block[6];
  block[6] = block[14];
  block[14] = tmp;

  //shift row 3 left by 3
  tmp = block[3];
  block[3] = block[15];
  block[15] = block[11];
  block[11] = block[7];
  block[7] = tmp;

}


//aes 128 encryption with expanded key supplied
//implemented as basic byte algorithm (naive)
//operates on one block per thread
__device__ void encrypt(uint8_t *block, uint8_t *rkey, uint32_t offset){

  add_round_key(block, rkey, offset);

  for(int i = 1; i < 10; ++i){
    sub_bytes(block, offset);
    shift_rows(block, offset);
    mix_columns(block, offset);
    add_round_key(block, rkey + 16*i, offset);
  }

  sub_bytes(block, offset);
  shift_rows(block, offset);
  add_round_key(block, rkey + 160, offset);

}



//aes 128 encryption with expanded key supplied
//implemented using 4 t-tables and sbox
//(watch for endianness) (1 block per thread)
__device__ void encrypt_full_table(uint8_t *block, uint8_t *rkey, uint32_t offset){
  
  uint8_t *b = (block + offset);
  uint32_t *bword = (uint32_t *)(block + offset);

  add_round_key(block, rkey, offset);

  for(int i = 1; i < 10; ++i){

    uint32_t *ckey = (uint32_t *)(rkey + 16*i);

    uint32_t c1 = te0[b[0]]  ^ te1[b[5]]  ^ te2[b[10]] ^ te3[b[15]] ^ ckey[0];
    uint32_t c2 = te0[b[4]]  ^ te1[b[9]]  ^ te2[b[14]] ^ te3[b[3]]  ^ ckey[1];
    uint32_t c3 = te0[b[8]]  ^ te1[b[13]] ^ te2[b[2]]  ^ te3[b[7]]  ^ ckey[2];
    uint32_t c4 = te0[b[12]] ^ te1[b[1]]  ^ te2[b[6]]  ^ te3[b[11]] ^ ckey[3];

    bword[0] = c1;
    bword[1] = c2;
    bword[2] = c3;
    bword[3] = c4;

  }

  sub_bytes(block, offset);
  shift_rows(block, offset);
  add_round_key(block, rkey + 160, offset);
}



//aes 128 encryption with expanded key supplied
//implemented using 1 t-tables (with rotation) and sbox
//1 block per thread
__device__ void encrypt_one_table(uint8_t *block, uint8_t *rkey, uint32_t offset){

  uint8_t *b = (block + offset);
  uint32_t *bword = (uint32_t *)(block + offset);

  add_round_key(block, rkey, offset);

  for(int i = 1; i < 10; ++i){

    uint32_t *ckey = (uint32_t *)(rkey + 16*i);

    uint32_t c1 = te0[b[0]]   ^ (te0[b[5]]<<8 | te0[b[5]]>>24)    ^ (te0[b[10]]<<16 | te0[b[10]]>>16) ^ (te0[b[15]]<<24 | te0[b[15]]>>8) ^ ckey[0];
    uint32_t c2 = te0[b[4]]   ^ (te0[b[9]]<<8 | te0[b[9]]>>24)    ^ (te0[b[14]]<<16 | te0[b[14]]>>16) ^ (te0[b[3]]<<24 | te0[b[3]]>>8)   ^ ckey[1];
    uint32_t c3 = te0[b[8]]   ^ (te0[b[13]]<<8 | te0[b[13]]>>24)  ^ (te0[b[2]]<<16 | te0[b[2]]>>16)   ^ (te0[b[7]]<<24 | te0[b[7]]>>8)   ^ ckey[2];
    uint32_t c4 = te0[b[12]]  ^ (te0[b[1]]<<8 | te0[b[1]]>>24)    ^ (te0[b[6]]<<16 | te0[b[6]]>>16)   ^ (te0[b[11]]<<24 | te0[b[11]]>>8) ^ ckey[3];

    bword[0] = c1;
    bword[1] = c2;
    bword[2] = c3;
    bword[3] = c4;

  }

  sub_bytes(block, offset);
  shift_rows(block, offset);
  add_round_key(block, rkey + 160, offset);

}





//perform aes 128 encryption with either a single table or 4 tables
//offset is the location of the working block in block
//boffset is the column in the working block (0 to 3)
//operates on 1 word per thread
__device__ void encrypt_full_perword(uint8_t *block, uint8_t *rkey, uint32_t offset, uint8_t col){

  uint8_t *b = block + offset;
  uint32_t *bword = (uint32_t *)(block + offset);   //start of the block  
  uint32_t *rwkey = (uint32_t *)rkey;

  //perform add_round_key  performed on single column
  bword[col] = bword[col] ^ rwkey[col];
  for(int i = 1; i < 10; ++i){
    uint32_t *ckey = (uint32_t *)(rkey + 16*i);
    int j = col * 4;


    //multiple t table
   // uint32_t c = te0[b[j]]  ^ te1[b[(j+5)&0xf]]  ^ te2[b[(j+10)&0xf]] ^ te3[b[(j+15)&0xf]] ^ ckey[col];

    //single t table
    
    uint32_t t1 = te0[b[j]];
    uint32_t t2 = te0[b[(j+5)&0xf]];
    uint32_t t3 = te0[b[(j+10)&0xf]];
    uint32_t t4 = te0[b[(j+15)&0xf]];

    uint32_t c = t1 ^ (t2<<8 | t2>>24) ^ (t3<<16 | t3>>16) ^ (t4<<24 | t4>>8) ^ ckey[col];
  
    bword[col] = c;
  }


  //subbytes
  uint8_t v1 = sbox[b[(col*4 + 0)&0xf]];
  uint8_t v2 = sbox[b[(col*4 + 5)&0xf]];
  uint8_t v3 = sbox[b[(col*4 + 10)&0xf]];
  uint8_t v4 = sbox[b[(col*4 + 15)&0xf]];

  //__syncthreads();  should all move together so not a broblem
  
  b[col*4 + 0] = v1;
  b[col*4 + 1] = v2;
  b[col*4 + 2] = v3;
  b[col*4 + 3] = v4;


  //add last round key
  bword[col] ^= rwkey[col + 40];
}


//perform counter mode encryption on block
//operates on a single word per thread with no memory fragmeting
__device__ void ctr_encrypt_perword(uint8_t *block, uint8_t *rkey, uint8_t *rseed, uint8_t *shmem, uint32_t toffset, uint8_t shblk, uint8_t col){
  uint32_t *b = (uint32_t *)block;
  uint32_t *r = (uint32_t *)rseed;
  uint32_t *sh = (uint32_t *)shmem;

  sh[shblk*4 + col] = r[col] + (col == 0)*(toffset / 16);

  //perform encryption
  encrypt_full_perword(shmem, rkey, shblk * 16, col);

  //xor with data
  b[toffset/4 + col] ^= sh[shblk*4 + col];

}



//perform counter mode encryption on block
//naive/ one-table/ or full table mode can be chosen by commenting/uncommenting
//operates on a single block per thread
__device__ void ctr_encrypt(uint8_t *block, uint8_t *rkey, uint8_t *rseed, uint32_t boffset, uint32_t toffset){
  uint32_t *b = (uint32_t *)block;
  uint32_t *r = (uint32_t *)rseed;
  uint32_t addpt[4];
  uint8_t *ctr_block = (uint8_t *)addpt;

  //word size traversal
  for(int i = 0; i < 4; ++i){
    addpt[i] = r[i];
  }

  //add in counter value
  addpt[0] = addpt[0] + toffset/16;
  //encrypt_full_table(ctr_block, rkey, 0);
   encrypt_one_table(ctr_block, rkey, 0);
  //encrypt(ctr_block, rkey, 0);

  //word size traversal
  for(int i = 0; i < 4; ++i){
    b[boffset/4 + i] ^= addpt[i];
  }
}



//basic encryption kernel.  Unused for ctr mode encryption
__global__ void encrypt_k(uint8_t *data, uint8_t *rkey, uint32_t numblock){
  int bindex = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = bindex * 16;
  if(bindex >= numblock) return;
  encrypt_one_table(data, rkey, offset);
}


//Temo test helper
__device__ void inc_block(uint8_t *data, uint32_t offset){  
  //word size traversal
  uint32_t *dat = (uint32_t *)data;
  for(int i = 0; i < 4; ++i){
    dat[offset/4 + i] ^= dat[offset/4 + i];
  }
}



//perfrom counter encryption using a single thread per word with no memory fragmentation
__global__ void ctr_encrypt_nofrag_perword(uint8_t *data, uint8_t *rkey, uint8_t *rseed, uint32_t numblock){
  uint32_t cindex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x; //index into column
  uint32_t bindex = cindex/4;
  uint32_t offset = bindex * 16;
  uint8_t shblk = bindex % 16;
  uint8_t col = cindex % 4;

  //memory for performing the encryption
  __shared__ uint32_t shmem[64];

  if(bindex >= numblock)return;

  ctr_encrypt_perword(data, rkey, rseed, (uint8_t *)shmem, offset, shblk, col);

}


//perform ctr encryption with a single thread per block with no memory fragmentation
__global__ void ctr_encrypt_k_nofrag(uint8_t *data, uint8_t *rkey, uint8_t *rseed, uint32_t numblock){
  int bindex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  int toffset = bindex * 16;  
  if(bindex >= numblock) return;

  ctr_encrypt(data, rkey, rseed, toffset, toffset);
}


//perform ctr encryption with a single thread per block with memory fragmentation to 
//enable better memory access patterns
__global__ void ctr_encrypt_k_frag(uint8_t *data, uint8_t *rkey, uint8_t *rseed, uint32_t numblock){
  __shared__ uint8_t smem[64 * 20];  
  uint32_t *swmem = (uint32_t *)smem;
  uint32_t *wdata = (uint32_t *)data;

  int bindex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  int toffset = bindex * 16;
  int boffset = threadIdx.x * 20; //5 bytes for better memory access patterns
  if(bindex >= numblock) return;

  for(int i = 0; i < 4; ++i){                       //copy block data to memory
    swmem[boffset/4 + i] = wdata[toffset/4 + i];
  }

  ctr_encrypt(smem, rkey, rseed, boffset, toffset);

  for(int i = 0; i < 4; ++i){                       //copy block data to memory
    wdata[toffset/4 + i] = swmem[boffset/4 + i];
  }
}




//handles running the encryption on the gpu
//key expansion is performed off gpu snce it is sequential
void encrypt_cuda_io(const uint8_t *inparray, uint8_t *outarray, uint8_t *key, uint8_t *rseed, uint32_t numblock){
  
  uint32_t num_bytes = numblock * 16;
  
  uint8_t rkey[176];
  
  expand_key(key, rkey);


  uint32_t *ddata;
  uint32_t *drkey;
  uint32_t *drseed;
 
  hipMalloc(&ddata, sizeof(uint8_t) * num_bytes);
  hipMalloc(&drkey, sizeof(uint8_t) * 176);
  hipMalloc(&drseed, sizeof(uint8_t) * 16);

  double out_start_time = CycleTimer::currentSeconds();
  
  hipMemcpy(ddata, (uint32_t *)inparray, sizeof(uint8_t) * num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(drkey, (uint32_t *)rkey, sizeof(uint8_t) * 176, hipMemcpyHostToDevice);
  hipMemcpy(drseed, (uint32_t *)rseed, sizeof(uint8_t) * 16, hipMemcpyHostToDevice);



  uint32_t maxblock = numblock;


  printf("%10.10u, ", numblock);

  dim3 nblock((numblock + 32*64 - 1)/(32*64),128);

//  double avg = 0;
//  for(int i = 0; i < 50; ++i){  

  double in_start_time = CycleTimer::currentSeconds();

  //choose kernel to run
  ctr_encrypt_nofrag_perword<<<nblock, 64>>>((uint8_t *)ddata, (uint8_t *)drkey, (uint8_t *)drseed, numblock);
  //ctr_encrypt_k_nofrag<<<nblock, 64>>>((uint8_t *)ddata, (uint8_t *)drkey, (uint8_t *)drseed, numblock);
  //ctr_encrypt_k_frag<<<nblock, 64>>>((uint8_t *)ddata, (uint8_t *)drkey, (uint8_t *)drseed, numblock);


  hipDeviceSynchronize();
  
  double in_end_time = CycleTimer::currentSeconds();
  double in_duration = in_end_time - in_start_time;

//  avg += in_duration * 1000.0;
  printf("%.3f, ", in_duration * 1000.0);
  fflush(stdout);
 // }
  

  printf("\n");
 // printf("%.3f,\n", avg/50.0);
  





  hipMemcpy(outarray, ddata, sizeof(uint8_t) * num_bytes, hipMemcpyDeviceToHost);

  double out_end_time = CycleTimer::currentSeconds();

  //check for errors
  hipError_t errCode = hipPeekAtLastError();
  if(errCode != hipSuccess){
    fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
  }

  double out_duration = out_end_time - out_start_time;
  //printf("GPU Overall Out Time: %.3f ms\n\n", 1000.0 * out_duration);

  hipFree(ddata);
  hipFree(drkey);
  hipFree(drseed);

}







/*


void encrypt_cuda(uint8_t *data, uint8_t *key, uint8_t *rseed, uint32_t numblock){
  
  uint32_t num_bytes = numblock * 16;
  
  uint8_t rkey[176];
  
  expand_key(key, rkey);


  uint32_t *ddata;
  uint32_t *drkey;
  uint32_t *drseed;
 
  hipMalloc(&ddata, sizeof(uint8_t) * num_bytes);
  hipMalloc(&drkey, sizeof(uint8_t) * 176);
  hipMalloc(&drseed, sizeof(uint8_t) * 16);

  double out_start_time = CycleTimer::currentSeconds();
  
  hipMemcpy(ddata, (uint32_t *)data, sizeof(uint8_t) * num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(drkey, (uint32_t *)rkey, sizeof(uint8_t) * 176, hipMemcpyHostToDevice);
  hipMemcpy(drseed, (uint32_t *)rseed, sizeof(uint8_t) * 16, hipMemcpyHostToDevice);

  double in_start_time = CycleTimer::currentSeconds();

  ctr_encrypt_k<<<(numblock + 31)/32, 32>>>((uint8_t *)ddata, (uint8_t *)drkey, (uint8_t *)drseed, numblock);
  hipDeviceSynchronize();
  
  double in_end_time = CycleTimer::currentSeconds();

  hipMemcpy(data, ddata, sizeof(uint8_t) * num_bytes, hipMemcpyDeviceToHost);

  double out_end_time = CycleTimer::currentSeconds();

  //check for errors
  hipError_t errCode = hipPeekAtLastError();
  if(errCode != hipSuccess){
    fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
  }

  double in_duration = in_end_time - in_start_time;
  double out_duration = out_end_time - out_start_time;
  printf("GPU Overall In  Time: %.3f ms\n", 1000.0 * in_duration);
  printf("GPU Overall Out Time: %.3f ms\n\n", 1000.0 * out_duration);

  hipFree(ddata);
  hipFree(drseed);
  hipFree(drkey);

}
*/

//OLD STUFF TO SAVE


/*
__global__ void encrypt_k_io(uint8_t *inpblock, uint8_t *outblock, uint8_t *rkey, uint32_t numblock){

  int bindex = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = bindex * 16;

  if(bindex >= numblock) return;

  copy_block(inpblock, outblock, offset);

  encrypt_one_table(outblock, rkey, offset);

}


__global__ void ctr_encrypt_k_io(uint8_t *inpblock, uint8_t *outblock, uint8_t *rkey, uint8_t *rseed, uint32_t numblock){

  int bindex = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = bindex * 16;

  if(bindex >= numblock) return;

  copy_block(inpblock, outblock, offset);

  ctr_encrypt(outblock, rkey, rseed, offset);

}
*/
